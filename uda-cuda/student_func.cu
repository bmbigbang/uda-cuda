#include "hip/hip_runtime.h"
// Homework 1
// Color to Greyscale Conversion

//A common way to represent color images is known as RGBA - the color
//is specified by how much Red, Grean and Blue is in it.
//The 'A' stands for Alpha and is used for transparency, it will be
//ignored in this homework.

//Each channel Red, Blue, Green and Alpha is represented by one byte.
//Since we are using one byte for each color there are 256 different
//possible values for each color.  This means we use 4 bytes per pixel.

//Greyscale images are represented by a single intensity value per pixel
//which is one byte in size.

//To convert an image from color to grayscale one simple method is to
//set the intensity to the average of the RGB channels.  But we will
//use a more sophisticated method that takes into account how the eye 
//perceives color and weights the channels unequally.

//The eye responds most strongly to green followed by red and then blue.
//The NTSC (National Television System Committee) recommends the following
//formula for color to greyscale conversion:

//I = .299f * R + .587f * G + .114f * B

//Notice the trailing f's on the numbers which indicate that they are 
//single precision floating point constants and not double precision
//constants.

//You should fill in the kernel as well as set the block and grid sizes
//so that the entire image is processed.
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <string>

#include <opencv2\core\core.hpp>
#include <opencv2\highgui\highgui.hpp>
#include <opencv2\opencv.hpp>
#include <hip/hip_runtime.h>
#include <string>

cv::MatND imageRGBA;
cv::MatND imageGrey;

uchar4        *d_rgbaImage__;
unsigned char *d_greyImage__;


size_t numRows() { return imageRGBA.rows; } //return # of rows in the image
size_t numCols() { return imageRGBA.cols; } //return # of cols in the image

void preProcess(uchar4 **h_rgbaImage, unsigned char **h_greyImage,
	uchar4 **d_rgbaImage, unsigned char **d_greyImage,
	const std::string& filename);

void postProcess(const std::string& output_file);

void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
	unsigned char* const d_greyImage, size_t numRows, size_t numCols);

__global__
void rgba_to_greyscale(const uchar4* const rgbaImage,
	unsigned char* const greyImage,
	int numRows, int numCols)
{
	for (int row = 0; row < numRows; ++row) {
		for (int col = 0; col < numCols; ++col) {
			uchar4 rgba = rgbaImage[row * numCols + col];
			float final2 = .299f * rgba.x + .587f * rgba.y + .114f * rgba.z;
			greyImage[row * numCols + col] = final2;
		}
	}
	//TODO
	//Fill in the kernel to convert from color to greyscale
	//the mapping from components of a uchar4 to RGBA is:
	// .x -> R ; .y -> G ; .z -> B ; .w -> A
	//
	//The output (greyImage) at each pixel should be the result of
	//applying the formula: output = .299f * R + .587f * G + .114f * B;
	//Note: We will be ignoring the alpha channel for this conversion

	//First create a mapping from the 2D block and grid locations
	//to an absolute 2D location in the image, then use that to
	//calculate a 1D offset
}



void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
	unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{
	//You must fill in the correct sizes for the blockSize and gridSize
	//currently only one block with one thread is being launched
	const dim3 blockSize(numRows, 1, 1);  //TODO
	const dim3 gridSize(numCols, 1, 1);  //TODO
	rgba_to_greyscale <<< gridSize, blockSize >>> (d_rgbaImage, d_greyImage, numRows, numCols);

	hipDeviceSynchronize();
}


//return types are void since any internal error will be handled by quitting
//no point in returning error codes...
//returns a pointer to an RGBA version of the input image
//and a pointer to the single channel grey-scale output
//on both the host and device
void preProcess(uchar4 **inputImage, unsigned char **greyImage,
	uchar4 **d_rgbaImage, unsigned char **d_greyImage,
	const std::string &filename) {
	//make sure the context initializes ok

	cv::Mat image;
	image = cv::imread(filename.c_str(), CV_LOAD_IMAGE_COLOR);
	if (image.empty()) {
		std::cerr << "Couldn't open file: " << filename << std::endl;
		exit(1);
	}

	cv::cvtColor(image, imageRGBA, CV_BGR2RGBA);

	//allocate memory for the output
	imageGrey.create(image.rows, image.cols, CV_8UC1);

	//This shouldn't ever happen given the way the images are created
	//at least based upon my limited understanding of OpenCV, but better to check
	if (!imageRGBA.isContinuous() || !imageGrey.isContinuous()) {
		std::cerr << "Images aren't continuous!! Exiting." << std::endl;
		exit(1);
	}

	*inputImage = (uchar4 *)imageRGBA.ptr<unsigned char>(0);
	*greyImage = imageGrey.ptr<unsigned char>(0);

	const size_t numPixels = numRows() * numCols();
	//allocate memory on the device for both input and output
	hipMalloc(d_rgbaImage, sizeof(uchar4) * numPixels);
	hipMalloc(d_greyImage, sizeof(unsigned char) * numPixels);
	hipMemset(*d_greyImage, 0, numPixels * sizeof(unsigned char)); //make sure no memory is left laying around

																	//copy input array to the GPU
	hipMemcpy(*d_rgbaImage, *inputImage, sizeof(uchar4) * numPixels, hipMemcpyHostToDevice);

	d_rgbaImage__ = *d_rgbaImage;
	d_greyImage__ = *d_greyImage;
}

void postProcess(const std::string& output_file) {
	const int numPixels = numRows() * numCols();
	//copy the output back to the host
	hipMemcpy(imageGrey.ptr<unsigned char>(0), d_greyImage__, sizeof(unsigned char) * numPixels, hipMemcpyDeviceToHost);

	//output the image
	cv::imwrite(output_file.c_str(), imageGrey);

	//cleanup
	hipFree(d_rgbaImage__);
	hipFree(d_greyImage__);
}



int main(int argc, char **argv) {
	uchar4        *h_rgbaImage, *d_rgbaImage;
	unsigned char *h_greyImage, *d_greyImage;

	std::string input_file;
	std::string output_file;
	if (argc == 3) {
		input_file = std::string(argv[1]);
		output_file = std::string(argv[2]);
	}
	else {
		std::cerr << "Usage: ./hw input_file output_file" << std::endl;
		exit(1);
	}
	//load the image and give us our input and output pointers
	preProcess(&h_rgbaImage, &h_greyImage, &d_rgbaImage, &d_greyImage, input_file);

	//call the students' code
	your_rgba_to_greyscale(h_rgbaImage, d_rgbaImage, d_greyImage, numRows(), numCols());

	hipDeviceSynchronize();
	printf("\n");

	//check results and output the grey image
	postProcess(output_file);

	return 0;
}
